#include "hip/hip_runtime.h"
#include <stdio.h>
#include "gnuastro/gpu.h"

__global__ void mat_add_kernel(int *a, int *b, int *c, int n)
{
    int row = blockIdx.y * n + threadIdx.y;
    int col = blockIdx.x * n + threadIdx.x;
    if (row < n && col < n)
    {
        c[row * n + col] = a[row * n + col] + b[row * n + col];
    }

}

void 
gal_gpu_mat_add(int *a, int *b, int n)
{
    int *c;
    int size = n * n * sizeof(int);
    c = (int *)malloc(size);

    int *device_a, *device_b, *device_c;

    hipMalloc((void **)&device_a, size);
    hipMalloc((void **)&device_b, size);
    hipMalloc((void **)&device_c, size);

    hipMemcpy(device_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(device_b, b, size, hipMemcpyHostToDevice);



    dim3 dimBlock(32, 32);
    dim3 dimGrid(ceil(n / 32.0), ceil(n / 32.0));

    mat_add_kernel<<<dimGrid, dimBlock>>>(device_a, device_b, device_c, n);

    hipMemcpy(c, device_c, size, hipMemcpyDeviceToHost);

    hipFree(device_a);
    hipFree(device_b);
    hipFree(device_c);

    for(int i=0;i<n;i++)
    {
        for(int j=0;j<n;j++)
        {
            printf("%d ", c[i*n+j]);
        }
        printf("\n");
    }

    free(c);

}